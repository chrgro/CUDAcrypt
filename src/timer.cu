#include "timer.h"
#include "stdio.h"

bool timer_running = false;
hipEvent_t timer_start, timer_stop; 
float timer_time; 

void timerStart() {
	if (!timer_running) {
		timer_running = true;
		
		hipEventCreate(&timer_start); 
		hipEventCreate(&timer_stop); 
		hipEventRecord( timer_start, 0 ); 
		
	} else {
		printf("Invalid timer start/stop order\n");
		exit(-1);
	}

}

float timerStop() {
	if (timer_running) {
		timer_running = false;
		
		hipEventRecord( timer_stop, 0 ); 
		hipEventSynchronize( timer_stop ); 
		hipEventElapsedTime( &timer_time, timer_start, timer_stop ); 
		hipEventDestroy( timer_start ); 
		hipEventDestroy( timer_stop );
		
		return timer_time;
	} else {
		printf("No timer running \n");
		exit(-1);
	}
}