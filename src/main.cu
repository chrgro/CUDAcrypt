#include "stdio.h"
#include "aes.h"
#include "timer.h"


__device__ unsigned char cexpkey[11][16];

int main(int argc, char *argv[]) {
	// Clear old error messages
	hipGetLastError();

	/*Added by Richard for input output*/
	FILE *in_file, *out_file;
	int in_index = 0, out_index = 0; //the argument index corresponding to in/out
	const char* in_str = "-i";
	const char* out_str = "-o";
	
	for(int i = 0; i < argc-1; i++)
	{
		if(strcmp(argv[i], in_str) == 0)
		{
			in_index = i + 1;
			break;
		}
	}

	for(int i = 0; i < argc-1; i++)
	{
		if(strcmp(argv[i], out_str) == 0)
		{
			out_index = i + 1;
			break;
		}
	}
	
	if (in_index == 0 || out_index == 0) {
		printf("Incorrect input parameters!\nUsage: bin/cudacrypt -i <INPUTFILE> -o <OUTPUTFILE>\n");
		exit(-1);
	}
	
	in_file = fopen(argv[in_index], "rb");
	if (in_file == false) {
		printf("Error: Input file cannot be opened (check path)\n");
		exit(-1);
	}
	
	out_file = fopen(argv[out_index], "wb");	
	if (out_file == false) {
		printf("Error: Output file cannot be created \n");
		exit(-1);
	}
	
	unsigned char *data;
	int datasize;
	int pad;
	
	
	fseek(in_file, 0L, SEEK_END);
	datasize = ftell(in_file);
	fseek(in_file, 0L, SEEK_SET);
	if(datasize%128) //not divisible by 128
		pad = 1;
	else//datasize is divisible by 128
		pad = 0;
	int numbytes = ((datasize/128) + pad) * 128;
	data = (unsigned char*)malloc(numbytes * sizeof(unsigned char));
	fread(data, 1, numbytes, in_file);
	fclose(in_file);


	unsigned char expkey[11][16];
	unsigned char aeskey[16] = {0x2b ,0x7e ,0x15 ,0x16 ,0x28 ,0xae ,0xd2 ,0xa6 ,
						  0xab ,0xf7 ,0x15 ,0x88 ,0x09 ,0xcf ,0x4f ,0x3c};
						  
	keySchedule(aeskey, expkey);
	
	
	float time;
	timerStart();
	

	
	// Set up GPU memory
	unsigned char *cdata;
	unsigned char *cexpkey;
	hipMalloc ( &cdata, numbytes*sizeof(unsigned char));
	hipMalloc ( &cexpkey, 11*16*sizeof(unsigned char));
	hipMemcpy ( cdata, data, numbytes*sizeof(unsigned char), hipMemcpyHostToDevice );
	hipMemcpy ( cexpkey, expkey, 11*16*sizeof(unsigned char), hipMemcpyHostToDevice );
	
	

	time = timerStop();
	printf ("Elapsed memory transfer time: %fms\n", time);
	
	// Run
	dim3 dimGrid ( (numbytes/128)/256 );
	dim3 dimBlock ( 256 );
	
	
	timerStart();
	aes128_core<<<dimGrid, dimBlock>>>((unsigned char(*)[16])cexpkey, cdata);
	time = timerStop();
	printf("Encryption time: %fms \n", time);
	
	
	
	timerStart();
	unsigned char* newdata = (unsigned char*)malloc(numbytes*sizeof(unsigned char));
	hipMemcpy ( newdata, cdata, numbytes*sizeof(unsigned char), hipMemcpyDeviceToHost );
	time = timerStop();
	printf("Copy from device: %fms\n", time);
	
	
	timerStart();
	fwrite (newdata , 1 , datasize*sizeof(unsigned char) , out_file);
	fclose (out_file);

	free(data);
	time = timerStop();
	printf("Write to file and closing operations time: %f\n", time);
	
	int err = hipPeekAtLastError();
	if (err != 0) {
		printf("Some error occured with the CUDA device! Error code: %i\n", err);
	}
	
	return 0;
}