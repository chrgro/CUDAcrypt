// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010


#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void hello(char *a, int *b) 
{
        a[threadIdx.x] += b[threadIdx.x];
}


__global__
void otp(int *v, int *k) {

		v[threadIdx.x] = v[threadIdx.x] ^ (*k);
}


int main() {
		int vals[N] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};
		int key = -1;
		
		for (int i=0; i<N; i++) {
			printf("%x,",vals[i]);
		}
		printf("\n");
		
		// Timer initiate
		hipEvent_t start, stop; 
		float time; 
		hipEventCreate(&start); 
		hipEventCreate(&stop); 
		hipEventRecord( start, 0 ); 
		
		
		// Set up GPU memory
		int *cvals;
		int *ckey;
		hipMalloc ( (void**)&cvals, N*sizeof(int));
		hipMalloc ( (void**)&ckey, sizeof(int));
		hipMemcpy ( cvals, vals, N*sizeof(int), hipMemcpyHostToDevice );
		hipMemcpy ( ckey, &key, sizeof(int), hipMemcpyHostToDevice );
		
		// Timer stop, reinitialize
		hipEventRecord( stop, 0 ); 
		hipEventSynchronize( stop ); 
		hipEventElapsedTime( &time, start, stop ); 
		hipEventDestroy( start ); 
		hipEventDestroy( stop );
		hipEventCreate(&start); 
		hipEventCreate(&stop); 
		hipEventRecord( start, 0 ); 
		
		// Output
		printf ("Elapsed memory transfer time: %fms\n", time);
		
		// Run OTP
		dim3 dimBlock ( blocksize, 1 );
		dim3 dimGrid ( 1, 1 );
		otp<<<dimGrid, dimBlock>>>(cvals, ckey);
		
		// Timer stop, reinitialize
		hipEventRecord( stop, 0 ); 
		hipEventSynchronize( stop ); 
		hipEventElapsedTime( &time, start, stop ); 
		hipEventDestroy( start ); 
		hipEventDestroy( stop );
		hipEventCreate(&start); 
		hipEventCreate(&stop); 
		hipEventRecord( start, 0 ); 
		
		// Output
		printf ("Elapsed OTP action time: %fms\n", time);
		
		// Retrieve data
		hipMemcpy( vals, cvals, N*sizeof(int), hipMemcpyDeviceToHost );
		hipDeviceSynchronize();
		hipFree( cvals );
		
		// Timer stop
		hipEventRecord( stop, 0 ); 
		hipEventSynchronize( stop ); 
		hipEventElapsedTime( &time, start, stop ); 
		hipEventDestroy( start ); 
		hipEventDestroy( stop );
		
		// Output
		printf ("Elapsed memory writeback time: %fms\n", time);
		for (int i=0; i<N; i++) {
			printf("%x,",vals[i]);
		}
		printf("\n");

}

// Helloworld main
int hellomain()
{
        char a[N] = "Hello \0\0\0\0\0\0";
        int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

        char *ad;
        int *bd;
        const int csize = N*sizeof(char);
        const int isize = N*sizeof(int);

        printf("%s", a);

        hipMalloc( (void**)&ad, csize ); 
        hipMalloc( (void**)&bd, isize ); 
        hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
        hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
        
        dim3 dimBlock( blocksize, 1 );
        dim3 dimGrid( 1, 1 );
        hello<<<dimGrid, dimBlock>>>(ad, bd);
        hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
        hipFree( ad );
        
        printf("%s\n", a);
        return EXIT_SUCCESS;
}