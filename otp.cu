
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__
void otp(int *v, int *k) {

		v[threadIdx.x] = v[threadIdx.x] ^ (*k);
}

int main() {
		int vals[N] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};
		int key = -1;
		
		for (int i=0; i<N; i++) {
			printf("%x,",vals[i]);
		}
		printf("\n");
		
		// Timer initiate
		hipEvent_t start, stop; 
		float time; 
		hipEventCreate(&start); 
		hipEventCreate(&stop); 
		hipEventRecord( start, 0 ); 
		
		
		// Set up GPU memory
		int *cvals;
		int *ckey;
		hipMalloc ( (void**)&cvals, N*sizeof(int));
		hipMalloc ( (void**)&ckey, sizeof(int));
		hipMemcpy ( cvals, vals, N*sizeof(int), hipMemcpyHostToDevice );
		hipMemcpy ( ckey, &key, sizeof(int), hipMemcpyHostToDevice );
		
		// Timer stop, reinitialize
		hipEventRecord( stop, 0 ); 
		hipEventSynchronize( stop ); 
		hipEventElapsedTime( &time, start, stop ); 
		hipEventDestroy( start ); 
		hipEventDestroy( stop );
		hipEventCreate(&start); 
		hipEventCreate(&stop); 
		hipEventRecord( start, 0 ); 
		
		// Output
		printf ("Elapsed memory transfer time: %fms\n", time);
		
		// Run OTP
		dim3 dimBlock ( blocksize, 1 );
		dim3 dimGrid ( 1, 1 );
		otp<<<dimGrid, dimBlock>>>(cvals, ckey);
		
		// Timer stop, reinitialize
		hipEventRecord( stop, 0 ); 
		hipEventSynchronize( stop ); 
		hipEventElapsedTime( &time, start, stop ); 
		hipEventDestroy( start ); 
		hipEventDestroy( stop );
		hipEventCreate(&start); 
		hipEventCreate(&stop); 
		hipEventRecord( start, 0 ); 
		
		// Output
		printf ("Elapsed OTP action time: %fms\n", time);
		
		// Retrieve data
		hipMemcpy( vals, cvals, N*sizeof(int), hipMemcpyDeviceToHost );
		hipDeviceSynchronize();
		hipFree( cvals );
		
		// Timer stop
		hipEventRecord( stop, 0 ); 
		hipEventSynchronize( stop ); 
		hipEventElapsedTime( &time, start, stop ); 
		hipEventDestroy( start ); 
		hipEventDestroy( stop );
		
		// Output
		printf ("Elapsed memory writeback time: %fms\n", time);
		for (int i=0; i<N; i++) {
			printf("%x,",vals[i]);
		}
		printf("\n");

}